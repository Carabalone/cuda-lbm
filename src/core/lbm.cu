#include "hip/hip_runtime.h"
#include "core/lbm.cuh"
#include "core/lbm_constants.cuh"
#include "functors/includes.cuh"


// #define DEBUG_KERNEL
#define PERIODIC
#define PERIODIC_X
#define PERIODIC_Y


#ifdef D2Q9
    __constant__ float WEIGHTS[quadratures];
    __constant__ int C[quadratures * dimensions];
    __constant__ float vis;
    __constant__ float tau;
    __constant__ float omega;
    __constant__ int OPP[quadratures];

    __constant__ float M[quadratures * quadratures];
    __constant__ float M_inv[quadratures * quadratures];
    __constant__ float S[quadratures];
#endif

// -----------------------------------------------------------------------------------------------------
// -----------------------------------------------------------------------------------------------------
// ------------------------------------MACROSCOPICS-----------------------------------------------------
// -----------------------------------------------------------------------------------------------------
// -----------------------------------------------------------------------------------------------------

__global__ void macroscopics_kernel(float* f, float* rho, float* u, float* force) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= NX || y >= NY) return;

    int node = y * NX + x;
    LBM::macroscopics_node(f, rho, u, force, node);
    if (node == DEBUG_NODE) {
        DPRINTF("[macroscopics_kernel] Node %d: rho=%f, u=(%f, %f)\n",
                node, rho[node], u[2*node], u[2*node+1]);
    }
}

void LBM::macroscopics() {
    dim3 blocks((NX + BLOCK_SIZE - 1) / BLOCK_SIZE,
                (NY+BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    macroscopics_kernel<<<blocks, threads>>>(d_f, d_rho, d_u, d_force);
    checkCudaErrors(hipDeviceSynchronize());
}

__device__
void LBM::macroscopics_node(float* f, float* rho, float* u, float* force, int node) {
    rho[node]       = 0.0f;
    u[2 * node]     = 0.0f;
    u[2 * node + 1] = 0.0f;

    for (int i=0; i < quadratures; i++) { 
         // f[get_node_index(node, i)] = 
         rho[node]       += f[get_node_index(node, i)];
         u[2 * node]     += f[get_node_index(node, i)] * C[2 * i];
         u[2 * node + 1] += f[get_node_index(node, i)] * C[2 * i + 1];
    }

    u[2 * node]     += 0.5f * force[2 * node];
    u[2 * node + 1] += 0.5f * force[2 * node + 1];

    // printf("adding %f\n", (force[2 * node]));

    u[2 * node]     *= 1.0f / rho[node];
    u[2 * node + 1] *= 1.0f / rho[node];

    // if (node == 0) {
    //     printf("Node 0: rho=%.4f, ux=%.4f, uy=%.4f\n", 
    //           rho[node], u[2*node], u[2*node+1]);
    // }
}

// -----------------------------------------------------------------------------------------------------
// -----------------------------------------------------------------------------------------------------
// --------------------------------------STREAMING------------------------------------------------------
// -----------------------------------------------------------------------------------------------------
// -----------------------------------------------------------------------------------------------------

__device__
void LBM::stream_node(float* f, float* f_back, int node) {
    const int x = node % NX;
    const int y = node / NX;
    const int baseIdx = get_node_index(node, 0);

    for (int i=1; i < quadratures; i++) {
        int x_neigh = x + C[2*i];
        int y_neigh = y + C[2*i+1];


#ifndef PERIODIC

        if (x_neigh < 0 || x_neigh >= NX || y_neigh < 0 || y_neigh >= NY)
            continue;

#else
    #ifdef PERIODIC_X
        #ifndef PERIODIC_Y
        if (y_neigh < 0 || y_neigh >= NY)
            continue;
        #endif
        x_neigh = (x_neigh + NX) % NX;
    #endif
    #ifdef PERIODIC_Y
        #ifndef PERIODIC_X
        if (x_neigh < 0 || x_neigh >= NX)
            continue;
        #endif
        y_neigh = (y_neigh + NY) % NY;
    #endif
#endif

        const int idx_neigh = get_node_index(NX * y_neigh + x_neigh, i);
        float source_val = f[baseIdx + i];

        f_back[idx_neigh] = f[baseIdx + i];

        if (fabsf(f_back[idx_neigh]) > VALUE_THRESHOLD || f_back[idx_neigh] < -0.01f) {
            printf("[WARNING][stream_node] Pushing negative/large value: "
                "Node (x=%3d, y=%3d) is pushing f[%d]=% .6f in Dir %d to neighbor at (x=%3d, y=%3d)\n",
                x, y, i, f[baseIdx + i], i, x_neigh, y_neigh);
        }
    }
}

__global__ void stream_kernel(float* f, float* f_back) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= NX || y >= NY) return;

    int node = y * NX + x;

    if (node == DEBUG_NODE) {
        DPRINTF("[stream_kernel] Before streaming (node %d):\n", node);
        for (int i = 0; i < quadratures; i++) {
            DPRINTF("    f[%d] = %f\n", get_node_index(node, i), f[get_node_index(node, i)]);
        }
    }

    LBM::stream_node(f, f_back, node);

    if (node == DEBUG_NODE) {
        DPRINTF("[stream_kernel] After streaming (node %d):\n", node);
        for (int i = 0; i < quadratures; i++) {
            DPRINTF("    f_back[%d] = %f\n", get_node_index(node, i), f_back[get_node_index(node, i)]);
        }
    }
}

void LBM::stream() {
    dim3 blocks((NX + BLOCK_SIZE - 1) / BLOCK_SIZE,
                (NY+BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    stream_kernel<<<blocks, threads>>>(d_f, d_f_back);
    checkCudaErrors(hipDeviceSynchronize());

    // float* temp;
    // temp = d_f;
    // d_f = d_f_back;
    // d_f_back = temp;
}
