#include "hip/hip_runtime.h"
#include "IBM/IBMManager.cuh"

IBMBody create_cylinder(float cx, float cy, float r, int num_pts) {

    IBMBody body = {num_pts, nullptr, nullptr};
    body.points = new float[2*num_pts];
    body.velocities = new float[2*num_pts];

    float angle = 2 * M_PI / num_pts; float coord[2];
    for (int i=0; i < num_pts; i++) {
        coord[0] = cx + r*cos(i*angle);
        coord[1] = cy + r*sin(i*angle);

        body.points[2*i]   = coord[0];
        body.points[2*i+1] = coord[1];
        
        body.velocities[2*i]   = 0.0f;
        body.velocities[2*i+1] = 0.0f;
    }


    return body;
}

IBMBody create_sphere(float cx, float cy, float cz, float r, int n_theta, int n_phi) {
    if (n_phi < 2) n_phi = 2;
    if (n_theta < 3) n_theta = 3; // need at least a triangle

    int num_pts = 2 + (n_phi - 2) * n_theta;
    IBMBody body { num_pts, nullptr, nullptr };
    body.points    = new float[3 * num_pts];
    body.velocities= new float[3 * num_pts];

    float dtheta = 2.0f * M_PI / n_theta;
    float dphi   = M_PI    / (n_phi - 1);

    int idx = 0;
    // 1) North pole (φ=0)
    body.points[3*idx + 0] = cx;
    body.points[3*idx + 1] = cy + r;
    body.points[3*idx + 2] = cz;
    body.velocities[3*idx + 0] = 0;
    body.velocities[3*idx + 1] = 0;
    body.velocities[3*idx + 2] = 0;
    ++idx;

    // 2) Intermediate rings φ in (dphi, (n_phi-2)*dphi)
    for (int i = 1; i < n_phi - 1; ++i) {
        float phi = i * dphi;
        float sinφ = sinf(phi);
        float cosφ = cosf(phi);
        for (int j = 0; j < n_theta; ++j) {
            float theta = j * dtheta;
            float cosθ = cosf(theta), sinθ = sinf(theta);

            float x = cx + r * sinφ * cosθ;
            float y = cy + r * cosφ;
            float z = cz + r * sinφ * sinθ;

            body.points[3*idx + 0] = x;
            body.points[3*idx + 1] = y;
            body.points[3*idx + 2] = z;
            body.velocities[3*idx + 0] = 0;
            body.velocities[3*idx + 1] = 0;
            body.velocities[3*idx + 2] = 0;
            ++idx;
        }
    }

    // 3) South pole (φ=π)
    body.points[3*idx + 0] = cx;
    body.points[3*idx + 1] = cy - r;
    body.points[3*idx + 2] = cz;
    body.velocities[3*idx + 0] = 0;
    body.velocities[3*idx + 1] = 0;
    body.velocities[3*idx + 2] = 0;
    // idx == num_pts-1 here

    return body;
}

template <>
__global__
void interpolate_velocities_kernel<2>(float* points, float* u_ibm, float* rho_ibm, int num_pts,
                                   float* u_lbm, float* rho_lbm) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= num_pts) return;

    float px = points[get_pt_index(idx, 0)]; float py = points[get_pt_index(idx, 1)];
    float gx = floorf(px); float gy = floorf(py);

    float ux = 0.0f; float uy = 0.0f;
    float rho = 0.0f;

    for (int i=0; i < 2; i++) {
        for (int j=0; j < 2; j++) {
            int nx = gx + i;
            int ny = gy + j;

            if (nx >= NX || nx < 0 || ny >= NY || ny < 0)
                continue;

            float dx = px - nx; float dy = py - ny;
            float k = kernel2D(dx, dy);
            int u_idx = nx + ny * NX;

            // if (idx==8) {
            //     printf("  Node %d: (%d, %d)\n", i*2+j, nx, ny);
            //     printf("    dx=%.6f, dy=%.6f\n", dx, dy);
            //     printf("    kernel=%.6f\n", k);
            //     printf("    u[%d]=(%.6f, %.6f)\n", u_idx, u_lbm[2*u_idx], u_lbm[2*u_idx+1]);
            //     printf("    current vel: (%.4f, %.4f)\n", ux, uy);
            // }

            rho += k * rho_lbm[u_idx];
            ux  += k * u_lbm[get_vec_index(u_idx, 0)];
            uy  += k * u_lbm[get_vec_index(u_idx, 1)];
        }
    }

    u_ibm[get_lag_vec_index(idx, 0, num_pts)] = ux;
    u_ibm[get_lag_vec_index(idx, 1, num_pts)] = uy;
    rho_ibm[idx] = rho;

    if (fabsf(ux) > 0.1f || fabsf(uy) > 0.1f)
        printf("  Interpolated velocity for point %d (%.4f, %.4f): (%.4f, %.4f)\n\trho=%.4f\n", idx, px, py, ux, uy, rho);
}

template <>
__global__
void interpolate_velocities_kernel<3>(float* points, float* u_ibm, float* rho_ibm, int num_pts,
                                   float* u_lbm, float* rho_lbm) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= num_pts) return;

    float px = points[get_pt_index(idx, 0)]; float py = points[get_pt_index(idx, 1)]; float pz = points[get_pt_index(idx, 2)];
    float gx = floorf(px); float gy = floorf(py); float gz = floorf(pz);

    float ux = 0.0f; float uy = 0.0f; float uz = 0.0f;
    float rho = 0.0f;

    for (int i=0; i < 2; i++) {
        for (int j=0; j < 2; j++) {
            for (int k=0; k < 2; k++) {
                int nx = gx + i;
                int ny = gy + j;
                int nz = gz + k;

                if (nx >= NX || nx < 0 || ny >= NY || ny < 0 || nz >= NZ || nz < 0)
                    continue;

                float dx = px - nx; float dy = py - ny; float dz = pz - nz;
                float kernel = kernel3D(dx, dy, dz);
                int u_idx = nx + ny * NX + nz * NX * NY;

                // if (idx==8) {
                //     printf("  Node %d: (%d, %d)\n", i*2+j, nx, ny);
                //     printf("    dx=%.6f, dy=%.6f\n", dx, dy);
                //     printf("    kernel=%.6f\n", k);
                //     printf("    u[%d]=(%.6f, %.6f)\n", u_idx, u_lbm[2*u_idx], u_lbm[2*u_idx+1]);
                //     printf("    current vel: (%.4f, %.4f)\n", ux, uy);
                // }

                rho += kernel * rho_lbm[u_idx];
                ux  += kernel * u_lbm[get_vec_index(u_idx, 0)];
                uy  += kernel * u_lbm[get_vec_index(u_idx, 1)];
                uz  += kernel * u_lbm[get_vec_index(u_idx, 2)];
            }
        }
    }

    u_ibm[get_lag_vec_index(idx, 0, num_pts)] = ux;
    u_ibm[get_lag_vec_index(idx, 1, num_pts)] = uy;
    u_ibm[get_lag_vec_index(idx, 2, num_pts)] = uz;
    rho_ibm[idx] = rho;

    if (fabsf(ux) > 0.12f || fabsf(uy) > 0.12f || fabsf(uz) >= 0.12f)
        printf("  Interpolated velocity for point %d (%.4f, %.4f, %.4f): (%.4f, %.4f, %.4f)\n\trho=%.4f\n",
             idx, px, py, pz, ux, uy, uz, rho);
}

template <>
__global__
void spread_forces_kernel<2>(float* points, float* forces_lagrangian, int num_pts, float* forces_eulerian) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= num_pts) return;

    float px = points[get_pt_index(idx, 0)]; float py = points[get_pt_index(idx, 1)];
    float gx = floorf(px); float gy = floorf(py);

    float fx = 0.0f; float fy = 0.0f;

    for (int i=0; i<2; i++) {
        for (int j=0; j<2; j++) {
            int nx = gx + i;
            int ny = gy + j;

            if (nx >= NX || nx < 0 || ny >= NY || ny < 0)
                continue;

            float dx = px - (gx + i); float dy = py - (gy + j);
            float k = kernel2D(dx, dy);

            fx = k * forces_lagrangian[get_lag_vec_index(idx, 0, num_pts)];
            fy = k * forces_lagrangian[get_lag_vec_index(idx, 1, num_pts)];

            // if (fabsf(fx) > 0.5f || fabsf(fy) > 0.5f)
            //     printf("forces in IBM (adding): (%.4f, %.4f)\n", fx, fy);

            int node_idx = ny * NX + nx;
            atomicAdd(&forces_eulerian[get_vec_index(node_idx, 0)], fx);
            atomicAdd(&forces_eulerian[get_vec_index(node_idx, 1)], fy);
        }
    }
}

template <>
__global__
void spread_forces_kernel<3>(float* points, float* forces_lagrangian, int num_pts, float* forces_eulerian) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= num_pts) return;

    float px = points[get_pt_index(idx, 0)]; float py = points[get_pt_index(idx, 1)]; float pz = points[get_pt_index(idx, 2)];
    float gx = floorf(px); float gy = floorf(py); float gz = floorf(pz);

    float fx = 0.0f; float fy = 0.0f; float fz = 0.0f;

    for (int i=0; i<2; i++) {
        for (int j=0; j<2; j++) {
            for (int k=0; k<2; k++) {
                int nx = gx + i;
                int ny = gy + j;
                int nz = gz + k;

                if (nx >= NX || nx < 0 || ny >= NY || ny < 0 || nz >= NZ || nz < 0)
                    continue;

                float dx = px - (gx + i); float dy = py - (gy + j); float dz = pz - (gz + k);
                float kernel = kernel3D(dx, dy, dz);

                fx = kernel * forces_lagrangian[get_lag_vec_index(idx, 0, num_pts)];
                fy = kernel * forces_lagrangian[get_lag_vec_index(idx, 1, num_pts)];
                fz = kernel * forces_lagrangian[get_lag_vec_index(idx, 2, num_pts)];

                // if (fabsf(fx) > 0.5f || fabsf(fy) > 0.5f || fabsf(fz) > 0.5f)
                //     printf("forces in IBM (adding): (%.4f, %.4f, %.4f)\n", fx, fy, fz);

                int node_idx = nz * NX * NY + ny * NX + nx;
                atomicAdd(&forces_eulerian[get_vec_index(node_idx, 0)], fx);
                atomicAdd(&forces_eulerian[get_vec_index(node_idx, 1)], fy);
                atomicAdd(&forces_eulerian[get_vec_index(node_idx, 2)], fz);
            }
        }
    }
}
