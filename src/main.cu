#include <stdio.h>
#include "lbm.cuh"
#include <iostream>


void setup_cuda() {
    checkCudaErrors(hipSetDevice(0));
    int deviceId = 0;
    checkCudaErrors(hipGetDevice(&deviceId));

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));

    size_t gpu_free_mem, gpu_total_mem;
    checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));

    printf("CUDA information\n");
    printf("       using device: %d\n", deviceId);
    printf("               name: %s\n",deviceProp.name);
    printf("    multiprocessors: %d\n", deviceProp.multiProcessorCount);
    printf(" compute capability: %d.%d\n", deviceProp.major,deviceProp.minor);
    printf("      global memory: %.1f MiB\n", deviceProp.totalGlobalMem/1048576.0);
    printf("        free memory: %.1f MiB\n", gpu_free_mem/1048576.0);
    printf("\n");

}

int main(void) {
    setup_cuda();


    std::cout << viscosity_to_tau(0.8) << std::endl;

    LBM lbm; // idea is control from host and give args to the kernels for the device.
    lbm.allocate();
    
    // while (timestaps) {
    //     lbm.stream(); <
    //     lbm.update_macroscopic(); <
    //     lbm.calc_equilibrium() <
    //     lbm.collide(); <
    //     lbm.process_boundary();
    // }

    const int totalTimesteps = 10000;
    int t = 0;

    lbm.init();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    while (t < totalTimesteps) {
        hipEventRecord(start);

        lbm.macroscopics();
        lbm.compute_equilibrium();
        lbm.collide();
        lbm.stream();
        lbm.apply_boundaries();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        if (t % (totalTimesteps / 1/*0*/) == 0) {
            float progress = (t * 100.0f) / totalTimesteps;
            printf("Simulation progress: %.1f%% (timestep %d/%d)\n", progress, t, totalTimesteps);
        }
        if (t % 100 == 0)
            lbm.save_macroscopics(t);

        t++;
    }

    lbm.free();

    return 0;
}
