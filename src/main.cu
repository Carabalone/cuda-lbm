#include <stdio.h>
#include "lbm.cuh"
#include "functors/includes.cuh"
#include <iostream>

#define SCENARIO TAYLOR_GREEN

#if defined(SCENARIO) && SCENARIO == TAYLOR_GREEN
    #include "scenarios/taylorGreen2D/TaylorGreenScenario.cuh"
    using Scenario = TaylorGreenScenario;
#else
    #include "scenarios/poiseuille/PoiseuilleScenario.cuh"
    using Scenario = PoiseuilleScenario;
#endif


void setup_cuda() {
    checkCudaErrors(hipSetDevice(0));
    int deviceId = 0;
    checkCudaErrors(hipGetDevice(&deviceId));

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));

    size_t gpu_free_mem, gpu_total_mem;
    checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));

    printf("CUDA information\n");
    printf("       using device: %d\n", deviceId);
    printf("               name: %s\n",deviceProp.name);
    printf("    multiprocessors: %d\n", deviceProp.multiProcessorCount);
    printf(" compute capability: %d.%d\n", deviceProp.major,deviceProp.minor);
    printf("      global memory: %.1f MiB\n", deviceProp.totalGlobalMem/1048576.0);
    printf("        free memory: %.1f MiB\n", gpu_free_mem/1048576.0);
    printf("\n");

}

int main(void) {
    setup_cuda();

    std::cout << "Running " << Scenario::name() << " scenario" << std::endl;
    std::cout << "Viscosity: " << Scenario::viscosity 
              << ", Tau: " << Scenario::tau << std::endl;

    LBM lbm; // idea is control from host and give args to the kernels for the device.
    lbm.allocate();

    const int total_timesteps = 10000;
    const int save_int = 100;
    int t = 0;

    lbm.init<Scenario>();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    while (t < total_timesteps) {
        bool save = (t+1)%save_int == 0;
        hipEventRecord(start);

        lbm.increase_ts<Scenario>();

        lbm.stream();
        lbm.swap_buffers();

        lbm.apply_boundaries();

        lbm.macroscopics();
        lbm.compute_equilibrium();
        lbm.collide();


        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        if (t % (total_timesteps / 20) == 0) {
            float progress = (t * 100.0f) / total_timesteps;
            printf("Simulation progress: %.1f%% (timestep %d/%d)\n", progress, t, total_timesteps);
        }
        if (save) {
            lbm.save_macroscopics(t+1); // save macroscopics updates the data from GPU to CPU.
            if constexpr (Scenario::has_analytical_solution) {
                printf("%s[%d]: error, %.2f%%\n", 
                       Scenario::name(), t+1,
                       lbm.compute_error<Scenario>());
            }
        }

        t++;
    }

    lbm.free();

    return 0;
}
